#include "hip/hip_runtime.h"
#include "Util.h"

__global__ void dot_kernel(int* A, int* B, int* S)
{
	__shared__ int temp[THREADS_PER_BLOCK];
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	temp[threadIdx.x] = A[index] * B[index];

	//_syncThreads();

	if (0 == threadIdx.x)
	{
		int sum = 0;
		for (int i = 0; i < THREADS_PER_BLOCK; i++)
			sum += temp[i];

		//atomicAdd(S, sum);
	}
}

extern "C" int dot(int* A, int* B, int* S, int N)
{
	int *devPtrA, *devPtrB, *devPtrS;

	// Alocate
	cudaSafeCall( hipMalloc((void**)&devPtrA, N * sizeof(int)) ); 
	cudaSafeCall( hipMalloc((void**)&devPtrB, N * sizeof(int)) );
	cudaSafeCall( hipMalloc((void**)&devPtrS, sizeof(int)) );

	cudaSafeCall( hipMemcpy(devPtrA, A, N * sizeof(int), hipMemcpyHostToDevice) ); 
	cudaSafeCall( hipMemcpy(devPtrB, B, N * sizeof(int), hipMemcpyHostToDevice) );

	//dot_kernel<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(devPtrA, devPtrB, devPtrS);

	// copy from device to host
	cudaSafeCall( hipMemcpy(S, devPtrS, sizeof(int), hipMemcpyDeviceToHost) ); 

	// free device memory
	cudaSafeCall( hipFree(devPtrA) ); 
	cudaSafeCall( hipFree(devPtrB) ); 
	cudaSafeCall( hipFree(devPtrS) ); 

	return 0;
}