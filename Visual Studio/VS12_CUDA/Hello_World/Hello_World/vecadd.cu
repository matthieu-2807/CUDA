#include "hip/hip_runtime.h"
#include "Util.h"

__global__ void vecAdd_kernel(float* A, float* B, float* C)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	C[index] = A[index] + B[index];
}

extern "C" float vecAdd(float* A, float* B, float* C, int N) 
{
	int i;
	float *dev_A, *dev_B, *dev_C; /* create devices variables to use it in device memory */

	clock_t t1, t2;
	float rt;

	/* allocate device memory  */
	cudaSafeCall( hipMalloc((void**)&dev_A, N * sizeof(float)) );
	cudaSafeCall( hipMalloc((void**)&dev_B, N * sizeof(float)) );
	cudaSafeCall( hipMalloc((void**)&dev_C, N * sizeof(float)) );

	for (i = 0; i < N; i++)  /* generate random data */
	{
		A[i] = (float)rand();
		B[i] = (float)RAND_MAX - A[i];
	}

	/* timing */
	t1 = clock();

	/* Copy variables into device memory variables */
	cudaSafeCall( hipMemcpy(dev_A, A, N * sizeof(float), hipMemcpyHostToDevice) );
	cudaSafeCall( hipMemcpy(dev_B, B, N * sizeof(float), hipMemcpyHostToDevice) );

	/* call compute kernel */
	vecAdd_kernel<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_A, dev_B, dev_C);

	/* Copy device memory variables into variables */
	cudaSafeCall( hipMemcpy(C, dev_C, N * sizeof(float), hipMemcpyDeviceToHost) );

	/* timing */
	t2 = clock();

	rt = (double)(t2 - t1)/CLOCKS_PER_SEC;
	
	/* free allocated device memory */
	cudaSafeCall( hipFree(dev_A) );
	cudaSafeCall( hipFree(dev_B) );
	cudaSafeCall( hipFree(dev_C) );

	return rt;
}