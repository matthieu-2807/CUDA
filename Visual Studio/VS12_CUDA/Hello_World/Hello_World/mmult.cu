#include "hip/hip_runtime.h"
#include "Util.h"

__global__ void mmult_kernel(float* A, float* B, float* C, int N)
{
	float sum = 0;

	int i = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
	int j = blockIdx.y;

	for (int k = 0; k < N*N; ++k)
		sum += B[i + N*k] * C[k + N*j];

	A[i + N*j] = sum;
}

extern "C" float mmult(float* A, float* B, float* C, int N)
{
	float *devPtrA, *devPtrB, *devPtrC;

	// Allocate
	cudaSafeCall( hipMalloc((void**)&devPtrA, N * N * sizeof(float)) ); 
	cudaSafeCall( hipMalloc((void**)&devPtrB, N * N * sizeof(float)) );
	cudaSafeCall( hipMalloc((void**)&devPtrC, N * N * sizeof(float)) );

	cudaSafeCall( hipMemcpy(devPtrB, B, N * N * sizeof(float), hipMemcpyHostToDevice) ); 
	cudaSafeCall( hipMemcpy(devPtrC, C, N * N * sizeof(float), hipMemcpyHostToDevice) );

	// define grid and thread block sizes
	dim3 threads(THREADS_PER_BLOCK);
    dim3 grid(N/THREADS_PER_BLOCK, N);

	//mmult_kernel<<< grid, threads >>>(devPtrA, devPtrB, devPtrC, N);

	// copy from device to host
	cudaSafeCall( hipMemcpy(A, devPtrA, sizeof(float), hipMemcpyDeviceToHost) ); 

	// free device memory
	cudaSafeCall( hipFree(devPtrA) );
	cudaSafeCall( hipFree(devPtrB) );
	cudaSafeCall( hipFree(devPtrC) );

	return 0;
}