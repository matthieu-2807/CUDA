#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h>

#include "utils.h"

__global__ void vecAdd_kernel(real* A, real* B, real* C) 
{ 
    // threadIdx.x is a built-in variable provided by CUDA at runtime 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    C[i] = A[i] + B[i]; 
}

extern "C" float vecAdd(real* A, real* B, real* C, int N) 
{
    real *devPtrA;
    real *devPtrB; 
    real *devPtrC; 

    hipEvent_t start, stop;

    float rt;

    cudaSafeCall( hipMalloc((void**)&devPtrA, N * sizeof(real)) ); 
    cudaSafeCall( hipMalloc((void**)&devPtrB, N * sizeof(real)) ); 
    cudaSafeCall( hipMalloc((void**)&devPtrC, N * sizeof(real)) ); 

    /* timing code */
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    cudaSafeCall( hipMemcpy(devPtrA, A, N * sizeof(real), hipMemcpyHostToDevice) ); 
    cudaSafeCall( hipMemcpy(devPtrB, B, N * sizeof(real), hipMemcpyHostToDevice) ); 

    /* call compute kernel */
    /* vecAdd(N, A, B, C); */
    vecAdd_kernel<<<N/512, 512>>>(devPtrA, devPtrB, devPtrC);
    
    cudaSafeCall( hipMemcpy(C, devPtrC, N * sizeof(real),  hipMemcpyDeviceToHost) ); 

    /* timing */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&rt, start, stop);  /* in milliseconds */
    rt /= 1E3;  /* convert to seconds */

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    cudaSafeCall( hipFree(devPtrA) ); 
    cudaSafeCall( hipFree(devPtrB) ); 
    cudaSafeCall( hipFree(devPtrC) ); 

    return rt;
}

